#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""

__global__ void CUDADrawPixels(uchar4 *pixels)
{
	int y = threadIdx.x + blockIdx.x * 1000;
	pixels[y] = { (unsigned char)y/100,(unsigned char)y/1000,(unsigned char)y,(unsigned char)255 };
}

void RunCUDA(uchar4 *d_destinationBitmap, int imageWidth, int imageHeight)
{
	dim3 threads(1000, 1, 1);
	dim3 grid(100);

	CUDADrawPixels << <grid, threads >> >(d_destinationBitmap);

	getLastCudaError("CUDADrawPixels kernel execution failed.\n");
}
